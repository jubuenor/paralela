#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <omp.h>

// #define BLOCKS  512
#define NUMTHREADS 8192
#define ITERATIONS 2e09

long double init, _end;
long double total_time;

__global__ void calculatePi(double *piTotal, long int iterations, int totalThreads)
{
    long int initIteration, endIteration;
    long int i = 0;
    double piPartial;

    int index = (blockDim.x * blockIdx.x) + threadIdx.x;

    initIteration = (iterations / totalThreads) * index;
    endIteration = initIteration + (iterations / totalThreads) - 1;

    i = initIteration;
    piPartial = 0;

    do
    {
        piPartial = piPartial + (double)(4.0 / ((i * 2) + 1));
        i++;
        piPartial = piPartial - (double)(4.0 / ((i * 2) + 1));
        i++;
    } while (i < endIteration);

    piTotal[index] = piPartial;

    __syncthreads();
    if (index == 0)
    {
        for (i = 1; i < totalThreads; i++)
            piTotal[0] = piTotal[0] + piTotal[i];
    }
}

int main(int argc, char *argv[])
{
    int blocksPerGrid, threadsPerBlock, i, size;
    long int iterations;
    int totalThreads;
    double *h_pitotal, *d_pitotal;

    sscanf(argv[1], "%i", &blocksPerGrid);
    hipError_t err = hipSuccess;
    FILE *fp = fopen("results.txt", "a");
    size = sizeof(double) * NUMTHREADS;
    h_pitotal = (double *)malloc(size);
    if (h_pitotal == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    for (i = 0; i < NUMTHREADS; i++)
        h_pitotal[i] = 0.0;

    err = hipMalloc((void **)&d_pitotal, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_pitotal, h_pitotal, sizeof(double) * NUMTHREADS, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    init = omp_get_wtime();

    threadsPerBlock = NUMTHREADS / blocksPerGrid;
    totalThreads = blocksPerGrid * threadsPerBlock;
    iterations = ITERATIONS;
    printf("\nCUDA kernel launch with %d blocks of %d threads Total: %i\n", blocksPerGrid, threadsPerBlock, totalThreads);
    fprintf(fp, "%d \t", blocksPerGrid);
    fprintf(fp, "%d \t", n_threads);
    calculatePi<<<blocksPerGrid, threadsPerBlock>>>(d_pitotal, iterations, totalThreads);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_pitotal, d_pitotal, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_pitotal);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    _end = omp_get_wtime();
    total_time += _end - init;
    fprintf(fp, "%Lf \n", total_time);
    printf("Tiempo total: %Lf\n", total_time);

    printf("\n%.12f", *h_pitotal);
    // Free host memory

    free(h_pitotal);
    err = hipDeviceReset();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return 0;
}
