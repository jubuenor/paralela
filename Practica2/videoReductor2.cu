#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <opencv2/opencv.hpp>

using namespace cv;  // Use OpenCV's namespace
using namespace std;  // Use the standard namespace

// Declare global variables for performance timing
long double init, _end;
long double total_time;

__global__ void reduce(Math *videoFrames, Math *finalVideoFrames, int n_frames, int n_threads){
    if(blockIdx.x<n_frames){
        Mat newFrame = Mat::zeros(videoFrames[blockIdx.x].first.size() / 3, videoFrames[blockIdx.x].first.type());
        for (int i = 0; i < videoFrames[blockIdx.x].first.rows; i += 3*n_threads)
        {
            for (int j = 0; j < videoFrames[blockIdx.x].first.cols; j += 3)
            {

                double blue = 0;
                double green = 0;
                double red = 0;

                for (int ik = 0; ik < 3; ik++)
                {
                    for (int jk = 0; jk < 3; jk++)
                    {
                        blue += videoFrames[blockIdx.x].first.at<Vec3b>(i+threadIdx.x + ik, j + jk)[0];
                        green += videoFrames[blockIdx.x].first.at<Vec3b>(i+threadIdx.x + ik, j + jk)[1];
                        red += videoFrames[blockIdx.x].first.at<Vec3b>(i+threadIdx.x + ik, j + jk)[2];
                    }
                }

                red /= 9;
                green /= 9;
                blue /= 9;
                Vec3b color = Vec3b(blue, green, red);
                newFrame.at<Vec3b>((i+threadIdx.x) / 3, j / 3) = color;
            }
        }
        finalVideoFrames[videoFrames[blockIdx.x].second] = newFrame;
    }
}

int main(int argc, char *argv[])
{
    // Initialize input and output video file names and number of threads from command-line arguments
    string input = argv[1];
    string output = argv[2];
    int n_blocks = atoi(argv[3]);  // Convert the third argument to an integer for number of blocks
    int n_threads = atoi(argv[4]);  // Convert the third argument to an integer for number of threads

    cout << "Iniciando programa con " << n_blocks << " bloques..." << endl;
    cout << "Iniciando programa con " << n_threads << " hilos..." << endl;

    // Open a file to write the results
    FILE *fp = fopen("results.txt", "a");
    fprintf(fp, "Bloques: %d \n", n_blocks);

    // Open input video file using OpenCV's VideoCapture
    VideoCapture cap(input);
    // Check if the video file is successfully opened
    if (!cap.isOpened())
    {
        cout << "Error opening video stream or file" << endl;
        return 0;
    }

    // Retrieve properties of the input video
    int fps = cap.get(CAP_PROP_FPS);
    int fourcc = cap.get(CAP_PROP_FOURCC);
    int frameCount = cap.get(CAP_PROP_FRAME_COUNT);

    // Initialize output video file using OpenCV's VideoWriter
    VideoWriter video(output, fourcc, fps, Size(640, 360));

    // Declare an array to hold the final processed frames
    Mat finalVideoFrames[frameCount];

    // Start performance timing
    init = omp_get_wtime();

    int n_frame = 0;

    // Main loop to read and process video frames
    while (true)
    {
        // Declare a vector to hold frames and their corresponding frame numbers
        vector<pair<Mat, int>> videoFrames;

        // Read 'n_threads' number of frames into the vector
        for (int i = 0; i < n_blocks; i++)
        {
            Mat frame;
            cap >> frame;

            if (frame.empty())
            {
                break;
            }
            videoFrames.push_back({frame, n_frame});
            n_frame++;
        }

        // Exit the loop if no frames are read
        if (videoFrames.empty())
        {
            break;
        }

        // Enable OpenMP parallelization with the specified number of threads
        reduce<<<n_blocks,n_threads>>>(d_videoFrames, d_finalVideoFrames, videoFrames.size(), n_threads);
        // Loop through the vector to process each frame

    }

    // End performance timing and calculate total time
    _end = omp_get_wtime();
    total_time = _end - init;
    fprintf(fp, "- Tiempo total: %Lfs \n", total_time);
    cout << "Tiempo total: " << total_time << "s" << endl;
    cout << "Resultado guardado en results.txt" << endl;

    // Close the results file
    fclose(fp);

    // Write the final processed frames to the output video
    for (int i = 0; i < frameCount; i++)
    {
        video.write(finalVideoFrames[i]);
        char c = (char)waitKey(1);
        if (c == 27)
            break;
    }

    // Release video resources
    cap.release();
    video.release();
    return 0;
}
