#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include <omp.h>

#include <opencv2/opencv.hpp>

using namespace cv;  // Use OpenCV's namespace
using namespace std;  // Use the standard namespace

// Declare global variables for performance timing
long double init, _end;
long double total_time;

__global__ void reduce(int *videoFrames, int *finalVideoFrames, int n_frames,int n_blocks, int n_threads, int width, int height){
    // loop into the matrix rows, the rows will be managed by the threads
    for (int i = 0; i < height; i += n_threads) // step of 3*n_threads, a thread manages a 3X3 pixel grid
    {   // loop into the matrix columns, the columns will be managed by the blocks
        for (int j = 0; j < width; j += n_blocks) //step of 3*n_blocks a block manages n_trheadsx3 grid
        {
            //initialize the color variables
            double blue = 0;
            double green = 0;
            double red = 0;
            //each pixel of the finalFrames will be a mean of a 3x3 grid of the original video frame
            for (int ik = 0; ik < 3; ik++)
            {
                for (int jk = 0; jk < 3; jk++)
                {
                    blue += videoFrames[(i+threadIdx.x)*27*width+(j+blockIdx.x)*9+9*jk+ik*9*width+0];  //sum over the blue value of the originals pixels
                    green += videoFrames[(i+threadIdx.x)*27*width+(j+blockIdx.x)*9+9*jk+ik*9*width+1]; //sum over the green value of the originals pixels
                    red += videoFrames[(i+threadIdx.x)*27*width+(j+blockIdx.x)*9+9*jk+ik*9*width+2];   //sum over the red value of the originals pixels
                }
            }
            //mean of the colors
            red /= 9;
            green /= 9;
            blue /= 9;

            finalVideoFrames[((int) ((i))+threadIdx.x)*width*3+ ((int) ((j))+blockIdx.x)*3+0] = blue;   //asign the neu color value to the final frame
            finalVideoFrames[((int) ((i))+threadIdx.x)*width*3+ ((int) ((j))+blockIdx.x)*3+1] = green;  //asign the neu color value to the final frame
            finalVideoFrames[((int) ((i))+threadIdx.x)*width*3+ ((int) ((j))+blockIdx.x)*3+2] = red;    //asign the neu color value to the final frame
        }
    }
}

int main(int argc, char *argv[])
{
    // Initialize input and output video file names and number of threads from command-line arguments
    string input = argv[1];
    string output = argv[2];
    int n_blocks = atoi(argv[3]);  // Convert the third argument to an integer for number of blocks
    int n_threads = atoi(argv[4]);  // Convert the third argument to an integer for number of threads

    cout << "Iniciando programa con " << n_blocks << " bloques..." << endl;
    cout << "Iniciando programa con " << n_threads << " hilos..." << endl;

    total_time = 0;

    // Open a file to write the results
    FILE *fp = fopen("results.txt", "a");
    fprintf(fp, "Bloques: %d \n", n_blocks);

    // Open input video file using OpenCV's VideoCapture
    VideoCapture cap(input);
    // Check if the video file is successfully opened
    if (!cap.isOpened())
    {
        cout << "Error opening video stream or file" << endl;
        return 0;
    }

    // Retrieve properties of the input video
    int fps = cap.get(CAP_PROP_FPS);
    int fourcc = cap.get(CAP_PROP_FOURCC);
    int frameCount = cap.get(CAP_PROP_FRAME_COUNT);

    // Initialize output video file using OpenCV's VideoWriter
    VideoWriter video(output, fourcc, fps, Size(640, 360));

    // Start performance timing

    hipError_t err = hipSuccess;
    int height = 360;
    int width = 640;

    //Memory allocation of the host input and output arrays
    int *finalVideoFrames = (int *)malloc(height*width*3*sizeof(int));
    int *videoFramesArray = (int *)malloc(3*height*3*width*3*sizeof(int));
    // Verify that allocations succeeded
    if (finalVideoFrames == NULL) {
        fprintf(stderr, "Failed to allocate host vector finalVideoFrames!\n");
        exit(EXIT_FAILURE);
    }

    if (videoFramesArray == NULL) {
        fprintf(stderr, "Failed to allocate host vector videoFramesArray!\n");
        exit(EXIT_FAILURE);
    }

    //Memory allocation  of the device array
    int *d_finalVideoFrames = NULL;
    err = hipMalloc((void **)&d_finalVideoFrames, height*width*3*sizeof(int));
    // Verify that dev allocations succeeded
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector videoFrames (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Memory allocation  of the device array
    int *d_videoFrames = NULL;
    err = hipMalloc((void **)&d_videoFrames, 3*height*3*width*3*sizeof(int));
    // Verify that dev allocations succeeded
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector videoFrames (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    printf("Iniciando procesamiento del video...\n");
    // Main loop to read and process video frames
    while (true)
    {
        // Declare a vector to hold frames and their corresponding frame numbers
        vector<pair<Mat, int>> videoFrames;
        int n_frame = 0;
        // Read 'n_threads' number of frames into the vector
        for (int i = 0; i < n_threads; i++)
        {
            Mat frame;
            cap >> frame;

            if (frame.empty())
            {
                break;
            }
            videoFrames.push_back({frame, n_frame});
            n_frame++;
        }

        // Exit the loop if no frames are read
        if (videoFrames.empty())
        {
            break;
        }

        //loop over the charged n_block frames of the original video
        for(int n =0; n<n_frame; n++){

            init = omp_get_wtime();
            //Copy the frame content to 1D Array
            for(int i = 0; i<height*3; i++){
                for(int j = 0; j<width*3; j++){
                    videoFramesArray[i*width*9+j*3+0] = videoFrames[n].first.at<Vec3b>(i, j)[0];
                    videoFramesArray[i*width*9+j*3+1] = videoFrames[n].first.at<Vec3b>(i, j)[1];
                    videoFramesArray[i*width*9+j*3+2] = videoFrames[n].first.at<Vec3b>(i, j)[2];
                }
            }
            _end = omp_get_wtime();
            total_time += _end - init;

            // Copy the content of the frame from Host to Device
            err = hipMemcpy(d_videoFrames, videoFramesArray, 3*height*3*width*3*sizeof(int), hipMemcpyHostToDevice);
            // Verify that copy succeeded
            if (err != hipSuccess) {
                fprintf(stderr, "Failed to copy vector videoFramesArray from host to device (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            // run cuda function
            reduce<<<n_blocks,n_threads>>>(d_videoFrames, d_finalVideoFrames, n_frame, n_blocks, n_threads, width, height);

            //Copy output data from the CUDA device to the host memory
            err = hipMemcpy(finalVideoFrames, d_finalVideoFrames, height*width*3*sizeof(int), hipMemcpyDeviceToHost);
            // Verify that copy succeeded
            if (err != hipSuccess) {
                fprintf(stderr, "Failed to copy vector finalVideoFrames from device to host (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }

            //Write the video

            //Create a new frame to allocate the new pixel's values
            Mat newFrame = Mat::zeros(videoFrames[n].first.size()/3, videoFrames[n].first.type());
            //loop over the pixel's values frame
            for(int i=0; i<height; i++){
                for(int j=0; j< width; j++){
                    int blue = finalVideoFrames[i*width*3+j*3+0];
                    int green = finalVideoFrames[i*width*3+j*3+1];
                    int red = finalVideoFrames[i*width*3+j*3+2];
                    Vec3b color = Vec3b(blue, green, red);
                    newFrame.at<Vec3b>(i, j) = color;
                }
            }

            // write the video
            video.write(newFrame);
            char c = (char)waitKey(1);
            if (c == 27)
                break;
        }
    }

    // End performance timing and calculate total time
    fprintf(fp, "- Tiempo total: %Lfs \n", total_time);
    cout << "Tiempo total: " << total_time << "s" << endl;
    cout << "Resultado guardado en results.txt" << endl;

    //clean the cuda memory
    err = hipFree(d_videoFrames);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_videoFrames (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //clean the cuda memory
    err = hipFree(d_finalVideoFrames);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_finalVideoFrames (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //clean the memory
    free(videoFramesArray);
    free(finalVideoFrames);
    // Close the results file
    fclose(fp);

    // Write the final processed frames to the output video

    // Release video resources
    cap.release();
    video.release();
    return 0;
}
