#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include <omp.h>

#include <opencv2/opencv.hpp>

using namespace cv;  // Use OpenCV's namespace
using namespace std; // Use the standard namespace

// Declare global variables for performance timing
long double init, _end;
long double total_time;

struct Pixel
{
    int red;
    int green;
    int blue;
};

__global__ void reduce(struct Pixel *videoFrames, struct Pixel *finalVideoFrames, int n_threads, int width, int size)
{
    int n = blockIdx.x;
    int a = 1080 / n_threads;
    int b = 1920 / n_threads;
    int thread = threadIdx.x;

    for (int i = thread * a; i < thread * a + a; i += 3)
    {
        for (int j = thread * b; j < thread * b + b; j += 3)
        {
            double blue = 0;
            double green = 0;
            double red = 0;

            for (int ik = 0; ik < 3; ik++)
            {
                for (int jk = 0; jk < 3; jk++)
                {
                    blue += videoFrames[n * size + i * width + j + ik * 3 + jk].blue;
                    green += videoFrames[n * size + i * width + j + ik * 3 + jk].green;
                    red += videoFrames[n * size + i * width + j + ik * 3 + jk].red;
                }
            }

            red /= 9;
            green /= 9;
            blue /= 9;
            finalVideoFrames[n * size + i / 3 * width / 3 + j / 3].blue = blue;
            finalVideoFrames[n * size + i / 3 * width / 3 + j / 3].green = green;
            finalVideoFrames[n * size + i / 3 * width / 3 + j / 3].red = red;

            // printf("(%d, %d, %d) ", finalVideoFrames[n * size + i * width / 3 + j].blue, finalVideoFrames[n * size + i * width / 3 + j].green, finalVideoFrames[n * size + i * width / 3 + j].red);
        }
        // printf("\n");
    }
}

int main(int argc, char *argv[])
{
    // Initialize input and output video file names and number of threads from command-line arguments
    string input = argv[1];
    string output = argv[2];
    int n_blocks = atoi(argv[3]);  // Convert the third argument to an integer for number of blocks
    int n_threads = atoi(argv[4]); // Convert the third argument to an integer for number of threads

    cout << "Iniciando programa con " << n_blocks << " bloques..." << endl;
    cout << "Iniciando programa con " << n_threads << " hilos..." << endl;

    // Open a file to write the results
    FILE *fp = fopen("results.txt", "a");
    fprintf(fp, "Bloques: %d \n", n_blocks);

    // Open input video file using OpenCV's VideoCapture
    VideoCapture cap(input);
    // Check if the video file is successfully opened
    if (!cap.isOpened())
    {
        cout << "Error opening video stream or file" << endl;
        return 0;
    }

    // Retrieve properties of the input video
    int fps = cap.get(CAP_PROP_FPS);
    int fourcc = cap.get(CAP_PROP_FOURCC);
    int frameCount = cap.get(CAP_PROP_FRAME_COUNT);

    // Initialize output video file using OpenCV's VideoWriter
    VideoWriter video(output, fourcc, fps, Size(640, 360));

    // Start performance timing
    init = omp_get_wtime();

    hipError_t err = hipSuccess;
    int initialHeight = 1080;
    int initialWidth = 1920;
    int finalHeight = 360;
    int finalWidth = 640;

    printf("Iniciando procesamiento del video...\n");
    // Main loop to read and process video frames
    while (true)
    {
        // Declare a vector to hold frames and their corresponding frame numbers
        vector<Mat> videoFrames;
        int n_frame = 0;
        // Read 'n_threads' number of frames into the vector
        for (int i = 0; i < n_blocks; i++)
        {
            Mat frame;
            cap >> frame;

            if (frame.empty())
            {
                break;
            }
            videoFrames.push_back(frame);
            n_frame++;
        }

        // Exit the loop if no frames are read
        if (videoFrames.empty())
        {
            break;
        }
        // Memory allocation of the input and output arrays
        int arraySizeInitial = videoFrames.size() * initialWidth * initialHeight * sizeof(struct Pixel);
        int arraySizeFinal = videoFrames.size() * finalWidth * finalHeight * sizeof(struct Pixel);

        struct Pixel *finalVideoFrames = (struct Pixel *)malloc(arraySizeFinal);
        struct Pixel *videoFramesArray = (struct Pixel *)malloc(arraySizeInitial);
        // Verify that allocations succeeded
        if (finalVideoFrames == NULL)
        {
            fprintf(stderr, "Failed to allocate host vector finalVideoFrames!\n");
            exit(EXIT_FAILURE);
        }

        if (videoFramesArray == NULL)
        {
            fprintf(stderr, "Failed to allocate host vector videoFramesArray!\n");
            exit(EXIT_FAILURE);
        }

        // Copy the frame content to 1D Array
        for (int n = 0; n < videoFrames.size(); n++)
            for (int i = 0; i < initialHeight; i++)
            {
                for (int j = 0; j < initialWidth; j++)
                {
                    videoFramesArray[n * videoFrames.size() + i * initialWidth + j].blue = videoFrames[n].at<Vec3b>(i, j)[0];
                    videoFramesArray[n * videoFrames.size() + i * initialWidth + j].green = videoFrames[n].at<Vec3b>(i, j)[1];
                    videoFramesArray[n * videoFrames.size() + i * initialWidth + j].red = videoFrames[n].at<Vec3b>(i, j)[2];
                    // printf("(%d, %d, %d)", videoFrames[n].at<Vec3b>(i, j)[0], videoFrames[n].at<Vec3b>(i, j)[1], videoFrames[n].at<Vec3b>(i, j)[2]);
                    // printf("(%d, %d, %d) ", videoFramesArray[n * videoFrames.size() + i * initialWidth + j].blue, videoFramesArray[n * videoFrames.size() + i * initialWidth + j].green, videoFramesArray[n * videoFrames.size() + i * initialWidth + j].red);
                }
                // printf("\n");
            }

        // Memory allocation  of the device array
        struct Pixel *d_videoFrames = NULL;
        err = hipMalloc((void **)&d_videoFrames, arraySizeInitial);
        // Verify that dev allocations succeeded
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector videoFrames (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Memory allocation  of the device array
        struct Pixel *d_finalVideoFrames = NULL;
        err = hipMalloc((void **)&d_finalVideoFrames, arraySizeFinal);
        // Verify that dev allocations succeeded
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector videoFrames (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Copy the content of the frame from Host to Device
        err = hipMemcpy(d_videoFrames, videoFramesArray, arraySizeInitial, hipMemcpyHostToDevice);
        // Verify that copy succeeded
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector videoFramesArray from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // run cuda function
        reduce<<<n_blocks, n_threads>>>(d_videoFrames, d_finalVideoFrames, n_threads, initialWidth, videoFrames.size());

        // copy output data from the cuda device to the host memory
        err = hipMemcpy(finalVideoFrames, d_finalVideoFrames, arraySizeFinal, hipMemcpyDeviceToHost);
        // verify that copy succeeded
        if (err != hipSuccess)
        {
            fprintf(stderr, "failed to copy vector finalvideoframes from device to host (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // write the video

        for (int n = 0; n < videoFrames.size(); n++)
        {
            Mat newFrame = Mat::zeros(videoFrames[n].size() / 3, videoFrames[n].type());
            for (int i = 0; i < finalHeight; i++)
            {
                for (int j = 0; j < finalWidth; j++)
                {
                    int blue = finalVideoFrames[n * videoFrames.size() + i * finalWidth + j].blue;
                    int green = finalVideoFrames[n * videoFrames.size() + i * finalWidth + j].green;
                    int red = finalVideoFrames[n * videoFrames.size() + i * finalWidth + j].red;
                    Vec3b color = Vec3b(blue, green, red);
                    newFrame.at<Vec3b>(i, j) = color;
                }
            }

            // for (int ik = 0; ik < newFrame.rows; ik++)
            //{
            // for (int jk = 0; jk < newFrame.cols; jk++)
            //{
            // printf("(%d, %d, %d) ", newFrame.at<Vec3b>(ik, jk)[0], newFrame.at<Vec3b>(ik, jk)[1], newFrame.at<Vec3b>(ik, jk)[2]);
            //}
            // printf("\n");
            //}
            video.write(newFrame);
        }

        char c = (char)waitKey(1);
        if (c == 27)
            break;
        hipFree(d_videoFrames);
        hipFree(d_finalVideoFrames);
        free(finalVideoFrames);
        free(videoFramesArray);
    }

    // End performance timing and calculate total time
    _end = omp_get_wtime();
    total_time = _end - init;
    fprintf(fp, "- Tiempo total: %Lfs \n", total_time);
    cout << "Tiempo total: " << total_time << "s" << endl;
    cout << "Resultado guardado en results.txt" << endl;

    // Close the results file
    fclose(fp);

    // Write the final processed frames to the output video

    // Release video resources
    cap.release();
    video.release();
    return 0;
}